#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <fstream>
#include <vector>
#include <iostream>
#include <cassert>

#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define size 6
#define MAX_RAY_DEPTH 3
const unsigned int window_width = 512;
const unsigned int window_height = 512;

struct vertexColor
{
        float x,y;
        float r,g,b;
};
struct Sphere{

	float3 color;
	float radius;
	float3 center;
	float3 emissionColor;
	float transparency;
	float reflection;

	__device__ bool hit(float3 origin, float3 direct, float *t0, float *t1) {
		float3 l = make_float3(center.x - origin.x, center.y-origin.y, center.z-origin.z);
		float tca = l.x*direct.x + l.y*direct.y+ l.z*direct.z;
		if(tca <0) return false;
		float d2 = l.x*l.x + l.y*l.y + l.z*l.z - tca*tca;
		if(d2 > radius*radius) return false;
		float thc = sqrtf(radius*radius - d2);
		*t0 = tca -thc;
                *t1 = tca +thc;
		return true;
	}
};

__constant__ Sphere s[size];

__device__ void normalize(float3 &f)
{
	float temp = sqrtf(f.x*f.x + f.y*f.y + f.z*f.z);
	f.x = f.x/temp; 
	f.y = f.y/temp;
	f.z = f.z/temp;
}

__device__ float mix(float a, float b, float mix)
{
    return b * mix + a * (1 - mix);
}

__device__ float3 trace(float3 origin, float3 direct, int depth)
{
    float dist = INFINITY;
    int intersect_object = -1;
    for(unsigned i=0; i<size; i++){
	float t0 = INFINITY, t1 = INFINITY; 
	if(s[i].hit(origin, direct, &t0, &t1))
	{
	    if(t0<0) t0 = t1;
	    if(t0< dist){
 		dist = t0;
		intersect_object = i;
	    }
	}
     }

    if(intersect_object == -1){
  	float3 result = make_float3(2, 2, 2);
	return result;
    }
    float3 surfaceColor = make_float3(0,0,0);
    float3 phit = make_float3(origin.x + direct.x*dist,origin.y + direct.y*dist,origin.z + direct.z*dist);
    float3 nhit = make_float3(phit.x-s[intersect_object].center.x, phit.y-s[intersect_object].center.y, phit.z-s[intersect_object].center.z);
    normalize(nhit);

    float bias = 1e-4;
    bool inside = false;
    if( (direct.x*nhit.x+direct.y*nhit.y+direct.z*nhit.z) > 0){
	nhit.x = (-1)*nhit.x;
	nhit.y = (-1)*nhit.y;
	nhit.z = (-1)*nhit.z;
	inside = true;
    }
 
    if((s[intersect_object].transparency > 0 || s[intersect_object].reflection > 0 ) && depth < MAX_RAY_DEPTH)
    {
	float tmp = direct.x*nhit.x + direct.y*nhit.y + direct.z*nhit.z;
	float facingratio = (-1)*tmp;
	float fresneleffect = mix(pow(1-facingratio, 3), 1, 0.1);
	float3 refldir = make_float3(direct.x - 2*tmp*nhit.x, direct.y - 2*tmp*nhit.y, direct.z - 2*tmp*nhit.z);
	normalize(refldir);
	float3 newReflRay = make_float3(phit.x + nhit.x*bias, phit.y + nhit.y*bias, phit.z + nhit.z*bias);
	float3 reflection = trace(newReflRay, refldir, depth+1);
	float3 refraction = make_float3(0,0,0);
	if(s[intersect_object].transparency){
	    float ior = 1.1, eta;
	    if(inside) eta = ior; else eta = 1/ior;
	    float cosi = (-1)*(nhit.x*direct.x + nhit.y*direct.y + nhit.z*direct.z);
	    float k = 1 -eta*eta*(1-cosi*cosi);
	    float3 refrdir = make_float3(direct.x*eta+nhit.x*(eta*cosi-sqrt(k)), direct.y*eta+nhit.y*(eta*cosi-sqrt(k)), direct.z*eta+nhit.z*(eta*cosi-sqrt(k)));
	    normalize(refrdir);
	    float3 newRefrRay = make_float3(phit.x-bias*nhit.x, phit.y-bias*nhit.y, phit.z-bias*nhit.z);
	    refraction = trace(newRefrRay, refrdir, depth+1);
	}

	surfaceColor.x = (reflection.x * fresneleffect + refraction.x * (1 - fresneleffect) * s[intersect_object].transparency )*s[intersect_object].color.x;
	surfaceColor.y = (reflection.y * fresneleffect + refraction.y * (1 - fresneleffect) * s[intersect_object].transparency )*s[intersect_object].color.y;
	surfaceColor.z = (reflection.z * fresneleffect + refraction.z * (1 - fresneleffect) * s[intersect_object].transparency )*s[intersect_object].color.z;
     } 
    else{
    // diffuse 
    	for(unsigned i=0; i<size; i++){
	    if(s[i].emissionColor.x>0){
	    	float3 transmission = make_float3(1,1,1);
	    	float3 lightDirection = make_float3(s[i].center.x-phit.x, s[i].center.y-phit.y, s[i].center.z-phit.z);
	    	normalize(lightDirection);
	    	for(unsigned j=0; j< size; j++){
		    if(i!=j){
		    	float t0, t1;
		    	float3 newRay = make_float3(phit.x+bias*nhit.x, phit.y+bias*nhit.y, phit.z+bias*nhit.z);
		    	if(s[j].hit(newRay, lightDirection, &t0, &t1)){
			    transmission.x = 0; transmission.y=0; transmission.z=0;
			    break;
		    	}
                    }  
                }
	        float tmp = nhit.x*lightDirection.x+nhit.y*lightDirection.y+nhit.z*lightDirection.z;
	        if(tmp<0) tmp = 0;

	       surfaceColor.x = surfaceColor.x+ s[intersect_object].color.x*transmission.x*tmp*s[i].emissionColor.x;
	       surfaceColor.y = surfaceColor.y+ s[intersect_object].color.y*transmission.y*tmp*s[i].emissionColor.y;
	       surfaceColor.z = surfaceColor.z+ s[intersect_object].color.z*transmission.z*tmp*s[i].emissionColor.z;
	  }
       }
    }
 

   float3 result2 = make_float3(surfaceColor.x+s[intersect_object].emissionColor.x, 
					 surfaceColor.y+s[intersect_object].emissionColor.y,
					 surfaceColor.z+s[intersect_object].emissionColor.z);
   return result2;
}

__global__ void tracer_kernel(vertexColor *pos)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
   if(x < window_width && y < window_height)
  {
    //----------compute ray----------------/
    float window_ratio = window_width / float(window_height);
    float fov = 90;
    float angle = tan(fov * 0.5 * M_PI / 180.0);
    float u = (2.0 * (x+0.5)/(float)window_width -1) * angle * window_ratio;
    float v = (1 - 2.0 * (y+0.5) / (float) window_height) * angle;
   
    float3 origin = make_float3(0,0,0);
    float3 direct = make_float3(u,v,-1);
    normalize(direct);
    //----------compute ray--------------//

    float3 pixel = trace(origin, direct, 0);
    pos[x+window_width*y].x = u;
    pos[x+window_width*y].y = v;
    pos[x+window_width*y].r = pixel.x;
    pos[x+window_width*y].g = pixel.y;
    pos[x+window_width*y].b = pixel.z;
   
  }
}

void init_kernel()
{
  Sphere *temp_s = (Sphere*)malloc(sizeof(Sphere)*size);
  temp_s[0].center = make_float3(0, 0, -20);
  temp_s[0].radius = 4;
  temp_s[0].color = make_float3(1.00, 0.32, 0.36);
  temp_s[0].emissionColor = make_float3(0,0,0);
  temp_s[0].transparency = 0.5;
  temp_s[0].reflection = 1;

  temp_s[1].center = make_float3(0, 20, -30);
  temp_s[1].radius = 3;
  temp_s[1].color = make_float3(0, 0, 0);
  temp_s[1].emissionColor = make_float3(3, 3, 3);
  temp_s[1].transparency = 0;
  temp_s[1].reflection = 0;

  temp_s[2].center = make_float3(0, -10004, -20);
  temp_s[2].radius = 10000;
  temp_s[2].color = make_float3(0.2, 0.2, 0.2);
  temp_s[2].emissionColor = make_float3(0, 0, 0);
  temp_s[2].transparency = 0;
  temp_s[2].reflection = 0;

  temp_s[3].center = make_float3(5, -1, -15);
  temp_s[3].radius = 2;
  temp_s[3].color = make_float3(0.9, 0.76, 0.46);
  temp_s[3].emissionColor = make_float3(0, 0, 0);
  temp_s[3].transparency = 0;
  temp_s[3].reflection = 1;

  temp_s[4].center = make_float3(5, 0, -25);
  temp_s[4].radius = 3;
  temp_s[4].color = make_float3(0.65, 0.77, 0.97);
  temp_s[4].emissionColor = make_float3(0, 0, 0);
  temp_s[4].transparency = 0;
  temp_s[4].reflection = 1;

  temp_s[5].center = make_float3(-5.5, -0, -15);
  temp_s[5].radius = 3;
  temp_s[5].color = make_float3(0.9, 0.9, 0.9);
  temp_s[5].emissionColor = make_float3(0, 0, 0);
  temp_s[5].transparency = 0;
  temp_s[5].reflection = 1;
  for(int i=0; i<size; i++){
	printf("x,y,z: %f, %f, %f, radius: %f\n", temp_s[i].center.x, temp_s[i].center.y,temp_s[i].center.z,temp_s[i].radius);
  }
  size_t sz = size*sizeof(Sphere);
  hipMemcpyToSymbol(HIP_SYMBOL(s), temp_s, sz, size_t(0), hipMemcpyHostToDevice);
 
  free(temp_s);
}


static void error_callback(int error, const char* description)
{
    fprintf(stderr, "Error: %s\n", description);
}

static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, GLFW_TRUE);
}


GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

void launch_kernel(vertexColor *pos)
{
    dim3 block(8, 8, 1);
    dim3 grid(window_width / block.x, window_height / block.y, 1);
    tracer_kernel<<< grid, block>>>(pos);
}

void runCuda(struct hipGraphicsResource **vbo_resource)
{
        vertexColor *dptr;
        hipGraphicsMapResources(1, vbo_resource, 0);
        size_t num_bytes;
        hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource);
        launch_kernel(dptr);

        hipGraphicsUnmapResources(1, vbo_resource, 0);
}

void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res, unsigned int vbo_res_flags)
{
        //Create vertex buffer object
        glGenBuffers(1, vbo);
        glBindBuffer(GL_ARRAY_BUFFER, *vbo);
        //Initialize VBO
        glBufferData(GL_ARRAY_BUFFER, window_width*window_height*sizeof(vertexColor), 0, GL_DYNAMIC_DRAW);
        glBindBuffer(GL_ARRAY_BUFFER, 0);
        //Register VBO with CUDA
        hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags);
}

int main(void)
{   
    //------ InitGL---------------//
    GLFWwindow* window;

    glfwSetErrorCallback(error_callback);

    if (!glfwInit())
        exit(EXIT_FAILURE);

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);

    window = glfwCreateWindow(window_width, window_height, "Simple example", NULL, NULL);
    if (!window)
    {   
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
    glfwSetKeyCallback(window, key_callback);

    glfwMakeContextCurrent(window);
    glewInit();
    glfwSwapInterval(1);

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    glViewport(0,0,window_width, window_height);
    //----------InitGL--------------//

    cudaGLSetGLDevice(0);

    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

    init_kernel();
    runCuda(&cuda_vbo_resource);

    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(2, GL_FLOAT, sizeof(float)*5, 0);
    glColorPointer(3, GL_FLOAT,sizeof(float)*5, (GLvoid*)(sizeof(float)*2));
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);

    while (!glfwWindowShouldClose(window))
    {
        glClear(GL_COLOR_BUFFER_BIT);

        glDrawArrays(GL_POINTS, 0, window_width * window_height);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    glfwDestroyWindow(window);

    glfwTerminate();
    exit(EXIT_SUCCESS);
}

